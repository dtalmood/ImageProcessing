#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
using namespace cv;
using namespace std;

// Custom comparator function for sorting pixels based on RGB values
bool comparePixels(const Vec3b& pixel1, const Vec3b& pixel2);

// Declaration of sortPixels function
extern "C" void sortPixels(uchar3* pixels, int numPixels);

int main() {
    // Load JPEG image using OpenCV
    // Read the image file
    Mat image = imread("source.jpeg");

    // Check if the image was successfully loaded
    if (image.empty()) {
        cout << "Could not open or find the image" << endl;
        return -1;
    }

    // Extract pixel values into a vector
    vector<Vec3b> pixels;
    for (int y = 0; y < image.rows; y++) {
        for (int x = 0; x < image.cols; x++) {
            pixels.push_back(image.at<Vec3b>(y, x));
        }
    }

    // Sort the pixels based on RGB values
    sortPixels(reinterpret_cast<uchar3*>(pixels.data()), pixels.size());

    // Create a new image with sorted pixels
    Mat sortedImage(image.size(), image.type());
    int index = 0;
    for (int y = 0; y < image.rows; y++) {
        for (int x = 0; x < image.cols; x++) {
            sortedImage.at<Vec3b>(y, x) = pixels[index++];
        }
    }

    imwrite("sortedPixels.jpeg", sortedImage);
    waitKey(0);

    return 0;
}

bool comparePixels(const Vec3b& pixel1, const Vec3b& pixel2) {
    if (pixel1[2] != pixel2[2]) {
        return pixel1[2] < pixel2[2]; // Sort by red channel
    }
    if (pixel1[1] != pixel2[1]) {
        return pixel1[1] < pixel2[1]; // Sort by green channel
    }
    return pixel1[0] < pixel2[0]; // Sort by blue channel
}
