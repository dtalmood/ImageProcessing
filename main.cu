#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

//incompatibility issues with open cv, header was needed for compilation
#include "kernel.h"

using namespace cv;
using namespace std;

int main() {
    //use opencv to read image
    cout << "Welcome!\n Please type the name of the image you would like to sort!" << endl;
    
    cin >> 

    Mat image = imread("source.jpeg");
    Timer timer;

    // Check if the image was successfully loaded
    if (image.empty()) {
        cout << "cannot read image" << endl;
        return -1;
    }

    //issues with opencv 2D matrix and cuda
    //vector is used instead


    printf("\n Phase 1: Traverse the 2D matrix and extract each pixel"); fflush(stdout);
    startTime(&timer);
    vector<uchar3> pixels;
    //traverse 2D matrix to extract pixel
    for (int y = 0; y < image.rows; y++) {
        for (int x = 0; x < image.cols; x++) {
            Vec3b bgrPixel = image.at<Vec3b>(y, x);
            uchar3 rgbPixel = make_uchar3(bgrPixel[2], bgrPixel[1], bgrPixel[0]);
            pixels.push_back(rgbPixel);
        }
    }
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Allocate host variables ----------------------------------------------

    // HOST VARIABLES ARE DYNAMIC HERE

    //==========================================


    // Allocate device variables ---------------------------------------------

    //incompatibility issues with opencv and cuda uchar3 is the solution
    uchar3* d_pixels;
    hipMalloc((void**)&d_pixels, pixels.size() * sizeof(uchar3));


    // Copy host variables to device ------------------------------------------
    hipMemcpy(d_pixels, pixels.data(), pixels.size() * sizeof(uchar3), hipMemcpyHostToDevice);


    // Launch kernel ---------------------------
    printf("\n Phase 2: Launch the pixelSort Kernel"); fflush(stdout);
    startTime(&timer);
    pixelSort(d_pixels, pixels.size());
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------
    hipMemcpy(pixels.data(), d_pixels, pixels.size() * sizeof(uchar3), hipMemcpyDeviceToHost);

    //building 2D opencv image
    Mat sortedImage(image.size(), image.type());
    for (int y = 0; y < image.rows; y++) {
        for (int x = 0; x < image.cols; x++) {

            uchar3 rgbPixel = pixels[y * image.cols + x];
            Vec3b bgrPixel(rgbPixel.z, rgbPixel.y, rgbPixel.x);
            sortedImage.at<Vec3b>(y, x) = bgrPixel;
        }
    }

    // Save the sorted image
    imwrite("sortedPixels.jpeg", sortedImage);

  
    hipFree(d_pixels);

    return 0;
}
