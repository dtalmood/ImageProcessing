#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <ctime>
//incompatibility issues with open cv, header was needed for compilation
#include "kernel.h"

using namespace cv;
using namespace std;
Mat importImage();

int main() 
{
    // We call the importImage Function which 
    Mat image = importImage();
    
    if (image.empty())
    {
        cout << "Goodbye!" << endl;
        return 0;
    } 
    else 
    {
	    time_t start;
        time_t current_time;

        // Check if the image was successfully loaded
        if (image.empty()) {
            cout << "cannot read image" << endl;
            return -1;
        }

        //issues with opencv 2D matrix and cuda
        //vector is used instead


        printf("\n Phase 1: Traverse the 2D matrix and extract each pixel\n"); fflush(stdout);
	    start = time(NULL);
        vector<uchar3> pixels;
        //traverse 2D matrix to extract pixel
        for (int y = 0; y < image.rows; y++) {
            for (int x = 0; x < image.cols; x++) {
                Vec3b bgrPixel = image.at<Vec3b>(y, x);
                uchar3 rgbPixel = make_uchar3(bgrPixel[2], bgrPixel[1], bgrPixel[0]);
                pixels.push_back(rgbPixel);
            }
        }
	    current_time = time(NULL);
	    int time_passed = current_time - start;
        int min = time_passed / 60;
        int sec = time_passed % 60;
        printf("It took %d minute(s) and %d second(s)\n", min, sec);

        // Allocate host variables ----------------------------------------------

        // HOST VARIABLES ARE DYNAMIC HERE

        //==========================================


        // Allocate device variables ---------------------------------------------

        //incompatibility issues with opencv and cuda uchar3 is the solution
        uchar3* d_pixels;
        hipMalloc((void**)&d_pixels, pixels.size() * sizeof(uchar3));


        // Copy host variables to device ------------------------------------------
        hipMemcpy(d_pixels, pixels.data(), pixels.size() * sizeof(uchar3), hipMemcpyHostToDevice);


        // Launch kernel ---------------------------
        printf("\n Phase 2: Launch the pixelSort Kernel\n"); fflush(stdout);
	    start = time(NULL);
        pixelSort(d_pixels, pixels.size());
	    current_time = time(NULL);
	    time_passed = current_time - start;
        min = time_passed / 60;
        sec = time_passed % 60;
        printf("It took %d minute(s) and %d second(s)\n", min, sec);

        // Copy device variables from host ----------------------------------------
        hipMemcpy(pixels.data(), d_pixels, pixels.size() * sizeof(uchar3), hipMemcpyDeviceToHost);

        //building 2D opencv image
        Mat sortedImage(image.size(), image.type());
        for (int y = 0; y < image.rows; y++) {
            for (int x = 0; x < image.cols; x++) {

                uchar3 rgbPixel = pixels[y * image.cols + x];
                Vec3b bgrPixel(rgbPixel.z, rgbPixel.y, rgbPixel.x);
                sortedImage.at<Vec3b>(y, x) = bgrPixel;
            }
        }

        // Save the sorted image
        imwrite("sortedPixels.jpeg", sortedImage);

        hipFree(d_pixels);
    }

    return 0;
}


Mat importImage() {
    string imageName;
    Mat image;

    while (true) {
        cout << "Please enter the name of the image you want to import (or 'q' to quit): ";
        cin >> imageName;

        if (imageName == "q") {
            cout << "Program terminated." << endl;
            break;
        }

        image = imread(imageName);

        if (image.empty()) {
            cout << "Image not found. Please try again." << endl;
        } else {
            cout << "Image imported successfully." << endl;
            return image;
        }
    }

    // Return an empty image if program is terminated or image not found
    return Mat();
}
