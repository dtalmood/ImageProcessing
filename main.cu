/*
     How to run code
        1. Make sure you are inside the directory of the file
        2. run command "make"
        3. run command "./imageprocessing"
*/
#include <stdio.h>
#include <stdint.h>
#include "kernel.cu"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <utility>
#include <vector>
#include <ctime>

using namespace cv;
using namespace std;

int main() {
    
    //Reading the images into 2D arrays w/ opencv =============================

    Mat reference = imread("source.jpeg"); //enter for image 1
    // Check for failure
    if (reference.empty()) {
        cout << "Could not open or find the image for 1" << endl;
        cin.get(); //wait for any key press
        return -1;
    }

    // Read the image file
    Mat source = imread("reference.jpeg");
    // Check for failure
    if (source.empty()) {
        cout << "Could not open or find the image for 2" << endl;
        cin.get(); //wait for any key press
        return -2;
    }
    if ((reference.rows != source.rows) || (reference.cols != source.cols)) {
        cout << "Images are not the same size" << endl;
        cin.get(); //wait for any key press
        return -3;
    }
    //==========================================================================



    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem...\n"); fflush(stdout);
    
    int size = reference.rows * reference.cols;
    Vec3b* ref_h, source_h, franken_h;
    Vec3b* ref_d, source_d, franken_d;
    
    ref_h = (Vec3b*)malloc(sizeof(Vec3b)*size);
    source_h = (Vec3b*)malloc(sizeof(Vec3b)*size);
    franken_h = (Vec3b*)malloc(sizeof(Vec3b)*size);
    

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables...\n"); fflush(stdout);

    hipMalloc((void**)&ref_d, size*sizeof(Vec3b));
    hipMalloc((void**)&source_d, size*sizeof(Vec3b));
    hipMalloc((void**)&franken_d, size*sizeof(Vec3b));



    hipDeviceSynchronize();

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device...\n"); fflush(stdout);
    
    hipMemcpy(ref_d, ref_h, size*sizeof(Vec3b), hipMemcpyHostToDevice);
    hipMemcpy(source_d, source_h, size*sizeof(Vec3b), hipMemcpyHostToDevice);



    hipDeviceSynchronize();

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel...\n"); fflush(stdout);
    
    frankenImage(ref_d, source_d, franken_d, size);
    cuda_ret = hipDeviceSynchronize();
	  if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host...\n"); fflush(stdout);
    
    hipMemcpy(franken_h, franken_d, size*sizeof(Vec3b), hipMemcpyDeviceToHost);


    hipDeviceSynchronize();
    //writing new image to image file =============================================
    //creating new image from source pixels using the reference image
    Mat frankenImage(reference.size(), reference.type());

    frankenImage = franken_d;

    imwrite("frankenImage.jpeg", frankenImage);

    
    // Free memory ------------------------------------------------------------

    free(ref_h);
    free(source_h);
    free(franken_h);

    hipFree(ref_d);
    hipFree(source_d);
    hipFree(franken_d);


    


    return 0;
}
