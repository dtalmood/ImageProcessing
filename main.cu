/*
     How to run code
        1. Make sure you are inside the directory of the file
        2. run command "make"
        3. run command "./imageprocessing"
*/
#include <stdio.h>
#include <stdint.h>

#include "kernel.cu"

int main() {
    
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem...\n"); fflush(stdout);
    

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables...\n"); fflush(stdout);



    hipDeviceSynchronize();

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device...\n"); fflush(stdout);



    hipDeviceSynchronize();

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel...\n"); fflush(stdout);
    
    

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host...\n"); fflush(stdout);
    


    hipDeviceSynchronize();
    
    // Free memory ------------------------------------------------------------



    return 0;
}
