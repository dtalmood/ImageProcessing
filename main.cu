#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include "support.h"
//incompatibility issues with open cv, header was needed for compilation
#include "kernel.h"

using namespace cv;
using namespace std;
Mat importImage();

int main() 
{
    // We call the importImage Function which 
    Mat image = importImage();
    
    if (importedImage.empty())
    {
        cout << "Goodbye!" << endl;
        return 0;
    } 
    else 
    {
        Timer timer;

        // Check if the image was successfully loaded
        if (image.empty()) {
            cout << "cannot read image" << endl;
            return -1;
        }

        //issues with opencv 2D matrix and cuda
        //vector is used instead


        printf("\n Phase 1: Traverse the 2D matrix and extract each pixel"); fflush(stdout);
        startTime(&timer);
        vector<uchar3> pixels;
        //traverse 2D matrix to extract pixel
        for (int y = 0; y < image.rows; y++) {
            for (int x = 0; x < image.cols; x++) {
                Vec3b bgrPixel = image.at<Vec3b>(y, x);
                uchar3 rgbPixel = make_uchar3(bgrPixel[2], bgrPixel[1], bgrPixel[0]);
                pixels.push_back(rgbPixel);
            }
        }
        stopTime(&timer); printf("%f s\n", elapsedTime(timer));

        // Allocate host variables ----------------------------------------------

        // HOST VARIABLES ARE DYNAMIC HERE

        //==========================================


        // Allocate device variables ---------------------------------------------

        //incompatibility issues with opencv and cuda uchar3 is the solution
        uchar3* d_pixels;
        hipMalloc((void**)&d_pixels, pixels.size() * sizeof(uchar3));


        // Copy host variables to device ------------------------------------------
        hipMemcpy(d_pixels, pixels.data(), pixels.size() * sizeof(uchar3), hipMemcpyHostToDevice);


        // Launch kernel ---------------------------
        printf("\n Phase 2: Launch the pixelSort Kernel"); fflush(stdout);
        startTime(&timer);
        pixelSort(d_pixels, pixels.size());
        stopTime(&timer); printf("%f s\n", elapsedTime(timer));

        // Copy device variables from host ----------------------------------------
        hipMemcpy(pixels.data(), d_pixels, pixels.size() * sizeof(uchar3), hipMemcpyDeviceToHost);

        //building 2D opencv image
        Mat sortedImage(image.size(), image.type());
        for (int y = 0; y < image.rows; y++) {
            for (int x = 0; x < image.cols; x++) {

                uchar3 rgbPixel = pixels[y * image.cols + x];
                Vec3b bgrPixel(rgbPixel.z, rgbPixel.y, rgbPixel.x);
                sortedImage.at<Vec3b>(y, x) = bgrPixel;
            }
        }

        // Save the sorted image
        imwrite("sortedPixels.jpeg", sortedImage);

        hipFree(d_pixels);
    }

    return 0;
}


Mat importImage() {
    string imageName;
    Mat image;

    while (true) {
        cout << "Please enter the name of the image you want to import (or 'q' to quit): ";
        cin >> imageName;

        if (imageName == "q") {
            cout << "Program terminated." << endl;
            break;
        }

        image = imread(imageName);

        if (image.empty()) {
            cout << "Image not found. Please try again." << endl;
        } else {
            cout << "Image imported successfully." << endl;
            return image;
        }
    }

    // Return an empty image if program is terminated or image not found
    return Mat();
}
