#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void sortPixels(Vec3b* pixelsReference, Vec3b* pixelsSource, pair<int, int>* position, int totalPixels) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= totalPixels)
        return;

    Vec3b swap;
    int currHue, tempHue;
    pair<int, int> tempPos;

    int counter, row, col;
    bool swapped;

    for (int k = 0; k < totalPixels; k++) {
        counter = 0;
        row = 0;
        col = 0;
        swapped = false;

        while (counter < totalPixels - k - 1) {
            currHue = pixelsReference[row * totalPixels + col][0];
            tempHue = pixelsReference[row * totalPixels + col + 1][0];
            if (currHue > tempHue) {
                swap = pixelsReference[row * totalPixels + col];
                pixelsReference[row * totalPixels + col] = pixelsReference[row * totalPixels + col + 1];
                pixelsReference[row * totalPixels + col + 1] = swap;
                swapped = true;
            }

            currHue = pixelsSource[row * totalPixels + col][0];
            tempHue = pixelsSource[row * totalPixels + col + 1][0];
            if (currHue > tempHue) {
                swap = pixelsSource[row * totalPixels + col];
                pixelsSource[row * totalPixels + col] = pixelsSource[row * totalPixels + col + 1];
                pixelsSource[row * totalPixels + col + 1] = swap;

                tempPos = position[row * totalPixels + col];
                position[row * totalPixels + col] = position[row * totalPixels + col + 1];
                position[row * totalPixels + col + 1] = tempPos;
                swapped = true;
            }

            col = (col + 1) % (totalPixels - 1);
            if (col == 0) {
                currHue = pixelsReference[row * totalPixels + totalPixels - 1][0];
                tempHue = pixelsReference[(row + 1) * totalPixels][0];
                if (row < totalPixels - 1 && currHue > tempHue) {
                    swap = pixelsReference[row * totalPixels + totalPixels - 1];
                    pixelsReference[row * totalPixels + totalPixels - 1] = pixelsReference[(row + 1) * totalPixels];
                    pixelsReference[(row + 1) * totalPixels] = swap;
                    swapped = true;
                }

                currHue = pixelsSource[row * totalPixels + totalPixels - 1][0];
                tempHue = pixelsSource[(row + 1) * totalPixels][0];
                if (row < totalPixels - 1 && currHue > tempHue) {
                    swap = pixelsSource[row * totalPixels + totalPixels - 1];
                    pixelsSource[row * totalPixels + totalPixels - 1] = pixelsSource[(row + 1) * totalPixels];
                    pixelsSource[(row + 1) * totalPixels] = swap;

                    tempPos = position[row * totalPixels + totalPixels - 1];
                    position[row * totalPixels + totalPixels - 1] = position[(row + 1) * totalPixels];
                    position[(row + 1) * totalPixels] = tempPos;
                    swapped = true;
                }
                counter++;
                row++;
            }
            counter++;
        }
        if (!swapped)
            break;
    }
}

void imageprocessing() {
    int blockSize = 256;
    int gridSize = (totalPixels + blockSize - 1) / blockSize;
    sortPixels<<<gridSize, blockSize>>>(d_pixelsReference, d_pixelsSource, d_position, totalPixels);
}