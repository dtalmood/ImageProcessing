#include "hip/hip_runtime.h"
#include "kernel.h"

__device__ bool sorter(const uchar3& pixel1, const uchar3& pixel2) {
    
    //red channel
    if (pixel1.z != pixel2.z) {
        return pixel1.z < pixel2.z; 
    }
    // green channel
    if (pixel1.y != pixel2.y) {
        return pixel1.y < pixel2.y; 
    }
    // Sort by blue channel
    return pixel1.x < pixel2.x; 
}

__global__ void sortPixels(uchar3* pixels, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        uchar3 currentPixel = pixels[idx];
        for (int i = idx + 1; i < size; i++) {
            uchar3 otherPixel = pixels[i];

            if (sorter(otherPixel, currentPixel)) {
                pixels[idx] = otherPixel;
                pixels[i] = currentPixel;
                currentPixel = pixels[idx];
            }
        }
    }
}

void pixelSort(uchar3* pixels, int size) {
    const unsigned int BLOCK_SIZE = 256;
    const unsigned int NUM_BLOCKS = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    sortPixels<<<NUM_BLOCKS, BLOCK_SIZE>>>(pixels, size);
    hipDeviceSynchronize();
}
