#include <hip/hip_runtime.h>

struct ComparePixel {
    __host__ __device__ bool operator()(const uchar3& pixel1, const uchar3& pixel2) const {
        if (pixel1.z != pixel2.z) {
            return pixel1.z < pixel2.z; // Sort by red channel
        }
        if (pixel1.y != pixel2.y) {
            return pixel1.y < pixel2.y; // Sort by green channel
        }
        return pixel1.x < pixel2.x; // Sort by blue channel
    }
};

extern "C" __global__ void sortPixels(uchar3* pixels, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        uchar3 currentPixel = pixels[idx];
        for (int i = idx + 1; i < size; i++) {
            uchar3 otherPixel = pixels[i];
            if (ComparePixel()(otherPixel, currentPixel)) {
                pixels[idx] = otherPixel;
                pixels[i] = currentPixel;
                currentPixel = pixels[idx];
            }
        }
    }

}
